
#include <hip/hip_runtime.h>
#include <device_atomic_functions.hpp>

// index trap as a 2D-array 
__device__ inline float idx(float *trap, int x, int y, int w) {
    return trap[(y*w)+x]; // y = row, x = col
    //return trap[(x*w)+y];
} 

// compute pixel value
__device__ inline float compute_pixel(float *trap, float omega, int x, int y, int w) {
	return (omega / 4.0) * (idx(trap, x-1, y  , w)
						 +  idx(trap, x+1, y  , w)
						 +  idx(trap, x  , y-1, w)
						 +  idx(trap, x  , y+1, w))
						 + (1.0 - omega) * idx(trap, x,y,w);
}


/*
 * simple
 */
 __global__ void simple(float *trap, int h, int w, float omega, float epsilon, int iter, float *delta) {

    // int row = blockDim.x * blockIdx.x + threadIdx.x;
	// int col = blockDim.y * blockIdx.y + threadIdx.y;
    // int trapIdx = (row*w)+col; // consider to scale, later

    int workPerBlock = (h*w)/gridDim.x;
    for (int idx = workPerBlock*blockIdx.x + threadIdx.x; idx < workPerBlock*blockIdx.x + workPerBlock; idx += blockDim.x){
        int x = (idx%w); 
        int y = (idx/w);
        if (x > 0 && x < (w-1) && y > 0 && y < (h-1)) { 
            float oldVal = trap[idx];
            float newVal = compute_pixel(trap, omega, x, y, w);
            trap[idx] = newVal;
            atomicAdd(delta, fabs(oldVal - newVal));
        }
    }
}


/*
 * rb
 */
__global__ void rb(float *trap, int h, int w, float omega, float epsilon, int total_iter, float *delta) {
    // implement me
    //printf("%d\n", total_iter);
    int workPerBlock = (h*w)/gridDim.x;
    for (int TrapIdx = workPerBlock*blockIdx.x + threadIdx.x; TrapIdx < workPerBlock*blockIdx.x + workPerBlock; TrapIdx += blockDim.x){
        int x = (TrapIdx%w); 
        int y = (TrapIdx/w);
        if (total_iter%2){
            if (((x%2==0) && (y%2==0)) || ((x%2==1) && (y%2==1))){
                if (x > 0 && x < (w-1) && y > 0 && y < (h-1)) { 
                    float oldVal = trap[TrapIdx];
                    float newVal = compute_pixel(trap, omega, x, y, w);
                    trap[TrapIdx] = newVal;
                    atomicAdd(delta, fabsf(oldVal - newVal));
                }
            }
        }
        else{
            if (((x%2==0) && (y%2==1)) || ((x%2==1) && (y%2==0))){
                if (x > 0 && x < (w-1) && y > 0 && y < (h-1)) { 
                    float oldVal = trap[TrapIdx];
                    float newVal = compute_pixel(trap, omega, x, y, w);
                    trap[TrapIdx] = newVal;
                    atomicAdd(delta, fabsf(oldVal - newVal));
                }
            }
        }
    }
}

/*
 * dbuf
 */
__global__ void dbuf(float *trap, float *trap2, int h, int w, float omega, float epsilon, int total_iter, float *delta) {
    int workPerBlock = (h*w)/gridDim.x;
    for (int idx = workPerBlock*blockIdx.x + threadIdx.x; idx < workPerBlock*blockIdx.x + workPerBlock; idx += blockDim.x){
        int x = (idx%w); 
        int y = (idx/w);
        if (total_iter%2){
            if (x > 0 && x < (w-1) && y > 0 && y < (h-1)) { 
                float oldVal = trap2[idx];
                float newVal = compute_pixel(trap2, omega, x, y, w);
                trap[idx] = newVal;
                atomicAdd(delta, fabs(oldVal - newVal));
            }
        }
        else{
            if (x > 0 && x < (w-1) && y > 0 && y < (h-1)) { 
                float oldVal = trap[idx];
                float newVal = compute_pixel(trap, omega, x, y, w);
                trap2[idx] = newVal;
                atomicAdd(delta, fabs(oldVal - newVal));
            }
        }
    }
}

